// Main
// Read in vertex, edge, and face data, allocate memory,
// create OpenGL context (through GLFW) and run the 
// simulations outlined in verlet.c
//
// Created by Lito Nicolai on 2/6/14.
// Copyright (c) 2014 Lito Nicolai. All rights reserved.
//
//
//
//
//

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#define GLFW_INCLUDE_GLU
#include <GLFW/glfw3.h>
#include "mesh_elements.h"
#include "verlet.h"
#include "draw.h"
#include "vert_export.h"
#include "bending_stiffness.h"

unsigned char paused = 0;
unsigned char momentum = 1;
unsigned char fixpt_on = 0;
unsigned char gravity_on = 0;

// Coefficient of Stiffness– lower is less stiff. 
// Keeping it < 1 is recommended. 0.1 is PLENTY stiff.
// 0 is completely floppy.
double stiffness_mu = 0.01;

// A lower value means slower and finer-grained simulation.
float simulation_speed = 0.1;

double currentTime = 0.0;
double lastTime = 0.0;
double deltaTime;

int main(int argc, const char * argv[])
{
    
    int NUM_PARTICLES;
    int NUM_EDGES;
    int NUM_FACEPAIRS;
    
    // Open verts file
    FILE *fv;
    fv = fopen("data/verts", "r+");

    if (fv == NULL){
        printf("Cannot find file ../data/verts\n");
        exit(-1);
    }

    // Get the number of verts
    fscanf(fv, "%d", &NUM_PARTICLES);

    // Allocate the particle-array
    Particle *p = (Particle *) calloc(NUM_PARTICLES,
            sizeof(Particle[NUM_PARTICLES]));
    
    // CUDA malloc
    Particle *device_p;
    hipMalloc(device_p, NUM_PARTICLES*sizeof(Particle));

    // Loop through the vert file to
    // fill the particle-array
    for (int i = 0; i < NUM_PARTICLES; i++){
        fscanf(fv, "%lf %lf %lf" 
                  "%lf %lf %lf", 
                &p[i].pos.x, &p[i].pos.y, &p[i].pos.z,
                &p[i].prev_pos.x, &p[i].prev_pos.y, &p[i].prev_pos.z);

        if (i == 0 && fixpt_on){
            p[i].invmass = 0.0;
        }
        else {
            p[i].invmass = 1.0;
        }
    }

    // Transfer results to CUDA
    hipMemcpy(device_p, p, NUM_EDGES*sizeof(Particle), hipMemcpyHostToDevice);

    fclose(fv);
    
    // Open edges file
    FILE *fe;
    fe = fopen("data/edges", "r+");

    if (fe == NULL){
        printf("Cannot find file ../data/edges\n");
        return -1;
    }

    // Get the number of edges
    fscanf(fe, "%d", &NUM_EDGES);

    // Allocate the edge-array
    Edge *e = (Edge *) calloc(NUM_EDGES,
            sizeof(Edge[NUM_EDGES]));
    
    // CUDA malloc
    Edge *device_e;
    hipMalloc(device_e, NUM_EDGES*sizeof(Edge));

    // Loop through the file to fill the 
    // edge-array
    for (int i = 0; i < NUM_EDGES; i++){
        fscanf(fe, "%d %d %lf",
                &e[i].a, &e[i].b, &e[i].length);
    }

    // Transfer results to CUDA
    hipMemcpy(device_e, e, NUM_EDGES*sizeof(Edge), hipMemcpyHostToDevice);

    fclose(fe);

    // Open facepairs file
    FILE *ffp;
    ffp = fopen("data/facepairs", "r+");

    if (ffp == NULL){
        printf("Cannot find file ../data/facepairs\n");
        return -1;
    }

    fscanf(ffp, "%d", &NUM_FACEPAIRS);

    printf("%d\n", NUM_FACEPAIRS);

    FacePair* fp = (FacePair *) calloc(NUM_FACEPAIRS,
            sizeof(FacePair[NUM_FACEPAIRS]));

    FacePair *device_fp;
    hipMalloc(device_fp, NUM_FACEPAIRS*sizeof(FacePair));

    for (int i = 0; i < NUM_FACEPAIRS; i++){
        fscanf(ffp, "%d %d %d %d",
                &fp[i].C, &fp[i].D, &fp[i].A, &fp[i].B);
        // Important! First two verts in a facepair are 
        // the internal edge, CD.
    }

    // Transfer results to CUDA
    hipMemcpy(device_fp, fp, NUM_FACEPAIRS*sizeof(FacePair), hipMemcpyHostToDevice);

    fclose(ffp);
    
    // Allocate memory for stiffness constants
    StiffnessDataContainer* StiffnessConstants = 
        (StiffnessDataContainer*) malloc(sizeof(StiffnessDataContainer));

    precompute_stiffness(stiffness_mu, StiffnessConstants);

        
/* ----------------------------------------------*/


    // OPENGL 
    


    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit()){
        return -1;
    }

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(640, 480, "Jellyfish Tentacles", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    glfwSetKeyCallback(window, key_callback);

    // Jitter once
    jitter_x(p, 0.5, NUM_PARTICLES);

    // Hide cursor
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_HIDDEN);

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        // TODO: FIX! 
        /*
        if (deltaTime < 0.0167){
            usleep((deltaTime - 0.0167)*1000000);
        }
        */

        currentTime = glfwGetTime();
        deltaTime = currentTime - lastTime;

        draw(window, deltaTime);

        glBegin(GL_LINES);
        for (int i = 0; i < NUM_EDGES; ++i){

            vector v1 = p[e[i].a].pos;
            vector v2 = p[e[i].b].pos;
            glColor3f(1.0, 1.0/v_magnitude(v_sub(v1, v2)), 0.0);

            glVertex3f(v1.x, v1.y, v1.z);
            glVertex3f(v2.x, v2.y, v2.z);
        }
        glEnd();

        // TEST AXES
        
        glBegin(GL_LINES);
        for (int i = 0; i < 3; ++i){
            vector v1 = {1.0, 0.0, 0.0};
            if (i == 1){
                v1.x = 0.0;
                v1.y = 1.0;
                v1.z = 0.0;
            }
            if (i == 2){
                v1.x = 0.0;
                v1.y = 0.0;
                v1.z = 1.0;
            }

            glColor3f(v1.x, v1.y, v1.z);
            glVertex3f(0.0, 0.0, 0.0);
            glVertex3f(v1.x, v1.y, v1.z);
        }
        glEnd();

       

        // Sticky Keys for paused
        // glfwSetInputMode(window, GLFW_STICKY_KEYS, GL_TRUE);

        // Check if pausedd
        if (glfwGetKey(window, GLFW_KEY_P ) == GLFW_PRESS){
            paused = ~paused;
        }

        // run the Verlet functions
        if (!paused) {

            if (gravity_on){
                    apply_gravity(p, NUM_PARTICLES, simulation_speed);
            }

            if (momentum){
                integrate_momentum(p, NUM_PARTICLES, deltaTime, simulation_speed);
            }
            runtime_stiffness(p, fp, NUM_FACEPAIRS, StiffnessConstants, simulation_speed);
            satisfy_constraints(p, e, NUM_EDGES, simulation_speed);
            resolve_collision(p, e, NUM_PARTICLES, simulation_speed);
        }

        lastTime = currentTime;

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    glfwTerminate();

    // Save the current vertex positions
    write_verts(p, e, NUM_PARTICLES, NUM_EDGES);

    return 0;
}

